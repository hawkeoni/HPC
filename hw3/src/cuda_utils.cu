
#include <hip/hip_runtime.h>
__global__ void calculate_inner_grid(double* grid_0, double* grid_1, double* grid_2, int bx, int by, int bz){
    int N = (bx + 2) * (by + 2) * (bz * 2);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i, j, k;
    double uijk = grid_1[idx], laplace = 0.;
    i = N % (bx + 2);
    if (i < 2 && i >= bx) return;
    j = N / (bx + 2) % (by + 2);
    if (j < 2 && j >= by) return;
    k = N / ((bx + 2) * (by + 2));
    if (k < 2 && k >= bz) return;
    grid_2[idx] = 2 * grid_1[idx] - grid_0[idx];
}

__global__ void first_step(double* grid_0, double* grid_1, \
        int bx, int by, int bz, \
        double hx, double hy, double hz, \
        double block_x_len, double block_y_len, double block_z_len, \
        int Lx, int Ly, int Lz, \
        int Nx, int Ny, int Nz, \
        int nx, int ny, int nz, \
        int block_pos_x, int block_pos_y, int block_pos_z,
        double at, double t){
    int N = (bx + 2) * (by + 2) * (bz * 2);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i, j, k;
    double uijk = grid_1[idx], laplace = 0.;
    double x, y, z;
    i = N % (bx + 2);
    if (i < 1 && i > bx) return;
    j = N / (bx + 2) % (by + 2);
    if (j < 1 && j > by) return;
    k = N / ((bx + 2) * (by + 2));
    if (k < 1 && k > bz) return;
    x = (i - 1) * hx + block_pos_x * block_x_len + min(Nx % nx, block_pos_x) * hx;
    y = (j - 1) * hy + block_pos_y * block_y_len + min(Ny % ny, block_pos_y) * hy;
    z = (k - 1) * hz + block_pos_z * block_z_len + min(Nz % nz, block_pos_z) * hz;
    grid_0[idx] = sin(3.14 / Lx * x) * sin(3.14 / Ly * y) * sin(2 * 3.14 / Lz * z) * cos(at * t + 2 * 3.14);
}
